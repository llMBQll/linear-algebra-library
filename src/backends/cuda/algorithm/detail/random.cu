#include "hip/hip_runtime.h"
#include "random.cuh"

#include <bit>
#include <complex>
#include <random>

namespace mbq::cuda::detail
{
    struct MT19937
    {
        hiprandGenerator_t generator{nullptr};

        MT19937()
        {
            hiprandCreateGenerator(&generator, hiprandRngType_t::HIPRAND_RNG_PSEUDO_MT19937);
            hiprandSetPseudoRandomGeneratorSeed(generator, std::random_device{}());
        }

        ~MT19937()
        {
            hiprandDestroyGenerator(generator);
        }
    };

    MT19937& get_default_engine()
    {
        thread_local static MT19937 engine;
        return engine;
    }

    template <>
    hiprandStatus_t random(float* ptr, size_t count, float /*min*/, float /*max*/)
    {
        auto& engine = get_default_engine();
        return hiprandGenerateUniform(engine.generator, ptr, count);
    }

    template <>
    hiprandStatus_t random(double* ptr, size_t count, double /*min*/, double /*max*/)
    {
        auto& engine = get_default_engine();
        return hiprandGenerateUniformDouble(engine.generator, ptr, count);
    }

    template <>
    hiprandStatus_t random(std::complex<float>* ptr, size_t count, std::complex<float> /*min*/,
                          std::complex<float> /*max*/)
    {
        auto& engine = get_default_engine();
        // cast to float and multiply count times 2 to account for real and imaginary parts
        return hiprandGenerateUniform(engine.generator, std::bit_cast<float*>(ptr), count * 2);
    }

    template <>
    hiprandStatus_t random(std::complex<double>* ptr, size_t count, std::complex<double> /*min*/,
                          std::complex<double> /*max*/)
    {
        auto& engine = get_default_engine();
        // cast to double and multiply count times 2 to account for real and imaginary parts
        return hiprandGenerateUniformDouble(engine.generator, std::bit_cast<double*>(ptr), count * 2);
    }
} // namespace mbq::cuda::detail
