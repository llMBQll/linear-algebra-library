#include "../../kernels/common.cuh"
#include "../../kernels/for_each.cuh"
#include "fill.cuh"

#include <complex>

namespace mbq::cuda::detail
{
    template <typename T>
    hipError_t fill(T* ptr, size_t count, T value)
    {
        return launch_for_each(cuda_cast(ptr), count, cuda_cast(value));
    }

    template hipError_t fill<float>(float* ptr, size_t count, float value);
    template hipError_t fill<double>(double* ptr, size_t count, double value);
    template hipError_t fill<std::complex<float>>(std::complex<float>* ptr, size_t count, std::complex<float> value);
    template hipError_t fill<std::complex<double>>(std::complex<double>* ptr, size_t count,
                                                    std::complex<double> value);
} // namespace mbq::cuda::detail